#include "hip/hip_runtime.h"
// Cuckoo Cycle, a memory-hard proof-of-work
// Copyright (c) 2013-2016 John Tromp

// The edge-trimming memory optimization is due to Dave Andersen
// http://da-data.blogspot.com/2014/03/a-public-review-of-cuckoo-cycle.html

#include <stdint.h>
#include <string.h>
#include "cuckoo.h"
#include "../crypto/siphash.cuh"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <set>

// algorithm parameters
#ifndef PART_BITS
// #bits used to partition edge set processing to save memory
// a value of 0 does no partitioning and is fastest
// a value of 1 partitions in two, making twice_set the
// same size as shrinkingset at about 33% slowdown
// higher values are not that interesting
#define PART_BITS 0
#endif

#ifndef IDXSHIFT
// we want sizeof(cuckoo_hash) == sizeof(twice_set), so
// CUCKOO_SIZE * sizeof(u64) == TWICE_WORDS * sizeof(u32)
// CUCKOO_SIZE * 2 == TWICE_WORDS
// (NNODES >> IDXSHIFT) * 2 == 2 * ONCE_BITS / 32
// NNODES >> IDXSHIFT == NEDGES >> PART_BITS >> 5
// IDXSHIFT == 1 + PART_BITS + 5
#define IDXSHIFT (PART_BITS + 6)
#endif

#define EDGEBITS 30
#define NODEBITS (EDGEBITS + 1)
#define NNODES (2 * NEDGES)
#define NODEMASK (NNODES-1)

// grow with cube root of size, hardly affected by trimming
#define MAXPATHLEN (8 << (NODEBITS/3))

#ifndef EDGE_BLOCK_BITS
#define EDGE_BLOCK_BITS 6
#endif
#define EDGE_BLOCK_SIZE (1 << EDGE_BLOCK_BITS)
#define EDGE_BLOCK_MASK (EDGE_BLOCK_SIZE - 1)
#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert (hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf (stderr, "GPUassert: %s %s %d\n", hipGetErrorString (code), file, line);
		if (abort)
			exit (code);
	}
}

// set that starts out full and gets reset by threads on disjoint words
class shrinkingset
{
  public:
	u32 * bits;
	__device__ void reset (word_t n)
	{
		bits[n / 32] |= 1 << (n % 32);
	}
	__device__ bool test (word_t n) const
	{
		return !((bits[n / 32] >> (n % 32)) & 1);
	}
	__device__ u64 block (word_t n) const
	{
		u64 r = *(u64 *) & bits[n / 32];
		//  return ~bits[n/32;
		  return ~r;
	}
};

#define PART_MASK ((1 << PART_BITS) - 1)
#define ONCE_BITS (NEDGES >> PART_BITS)
#define TWICE_WORDS ((2 * ONCE_BITS) / 32)

class twice_set
{
  public:
	u32 * bits;
	__device__ void reset ()
	{
		memset (bits, 0, TWICE_WORDS * sizeof (u32));
	}
	__device__ void set (word_t u)
	{
		word_t idx = u / 16;
		u32 bit = 1 << (2 * (u % 16));
		u32 old = atomicOr (&bits[idx], bit);
		u32 bit2 = bit << 1;
		if ((old & (bit2 | bit)) == bit)
			atomicOr (&bits[idx], bit2);
	}
	__device__ u32 test (word_t u) const
	{
		return (bits[u / 16] >> (2 * (u % 16))) & 2;
	}
};

#define CUCKOO_SIZE (NNODES >> IDXSHIFT)
#define CUCKOO_MASK (CUCKOO_SIZE - 1)
// number of (least significant) key bits that survives leftshift by NODEBITS
#define KEYBITS (64-NODEBITS)
#define KEYMASK ((1L << KEYBITS) - 1)
#define MAXDRIFT (1L << (KEYBITS - IDXSHIFT))

class cuckoo_hash
{
  public:
	u64 * cuckoo;

	cuckoo_hash ()
	{
		cuckoo = (u64 *) calloc (CUCKOO_SIZE, sizeof (u64));
		assert (cuckoo != 0);
	}
	 ~cuckoo_hash ()
	{
		free (cuckoo);
	}
	void set (word_t u, word_t v)
	{
		u64 niew = (u64) u << NODEBITS | v;
		for (word_t ui = u >> IDXSHIFT;; ui = (ui + 1) & CUCKOO_MASK)
		{
#ifdef ATOMIC
			u64 old = 0;
			if (cuckoo[ui].compare_exchange_strong (old, niew, std::memory_order_relaxed))
				return;
			if ((old >> NODEBITS) == (u & KEYMASK))
			{
				cuckoo[ui].store (niew, std::memory_order_relaxed);
#else
			u64 old = cuckoo[ui];
			if (old == 0 || (old >> NODEBITS) == (u & KEYMASK))
			{
				cuckoo[ui] = niew;
#endif
				return;
			}
		}
	}
	word_t operator[] (word_t u) const
	{
		for (word_t ui = u >> IDXSHIFT;; ui = (ui + 1) & CUCKOO_MASK)
		{
#ifdef ATOMIC
			u64 cu = cuckoo[ui].load (std::memory_order_relaxed);
#else
			u64 cu = cuckoo[ui];
#endif
			if (!cu)
				  return 0;
			if ((cu >> NODEBITS) == (u & KEYMASK))
			{
				assert (((ui - (u >> IDXSHIFT)) & CUCKOO_MASK) < MAXDRIFT);
				return (word_t) (cu & NODEMASK);
			}
		}
	}
};

// arbitrary length of header hashed into siphash key
#define HEADERLEN 80

class cuckoo_ctx
{
  public:
	siphash_keys sip_keys;
	shrinkingset alive;
	twice_set nonleaf;
	int nthreads;

	  cuckoo_ctx (const u32 n_threads)
	{
		nthreads = n_threads;
	}
	void setheadernonce (char *headernonce, const u32 nonce)
	{
		((u32 *) headernonce)[HEADERLEN / sizeof (u32) - 1] = htole32 (nonce);	// place nonce at end
		setheader (headernonce, HEADERLEN, &sip_keys);
	}
};

__device__ u64 dipblock (const siphash_keys & keys, const word_t edge, u64 * buf)
{
	u64 v0 = keys.k0, v1 = keys.k1, v2 = keys.k2, v3 = keys.k3;
	word_t edge0 = edge & ~EDGE_BLOCK_MASK;
	u32 i;
	for (i = 0; i < EDGE_BLOCK_MASK; i++)
	{
		word_t nonce = edge0 + i;
		v3 ^= nonce;
		SIPROUND;
		SIPROUND;
		v0 ^= nonce;
		v2 ^= 0xff;
		SIPROUND;
		SIPROUND;
		SIPROUND;
		SIPROUND;
		buf[i] = (v0 ^ v1) ^ (v2 ^ v3);
	}
	word_t nonce = edge0 + i;
	v3 ^= nonce;
	SIPROUND;
	SIPROUND;
	v0 ^= nonce;
	v2 ^= 0xff;
	SIPROUND;
	SIPROUND;
	SIPROUND;
	SIPROUND;

	buf[i] = 0;
	return (v0 ^ v1) ^ (v2 ^ v3);
}

/*
__global__ void count_node_deg(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys; // local copy sip context; 2.5% speed gain
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 block = id*64; block < NEDGES; block += ctx->nthreads*64) {
    u64 alive32 = alive.block(block);
    for (u32 nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffsll(alive32);
      nonce += ffs; alive32 >>= ffs;
      u32 u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        nonleaf.set(u >> PART_BITS);
      }
    }
  }
}*/

__global__ void count_node_deg (cuckoo_ctx * ctx, u32 uorv, u32 part, unsigned long long int *hash_count, int* nonce_hash_count, unsigned long long int *rw_count)
{
	shrinkingset & alive = ctx->alive;
	twice_set & nonleaf = ctx->nonleaf;
	siphash_keys sip_keys = ctx->sip_keys;	// local copy sip context; 2.5% speed gain
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	u64 buf[64];
	for (u32 block = id * 64; block < NEDGES; block += ctx->nthreads * 64)
	{
		u64 alive64 = alive.block (block);
		u64 last = 0;
		if(alive64) // if commenting this code, will more quickly
		{
			last = dipblock (sip_keys, block, buf);
	//		atomicAdd(hash_count, 64);
	//		atomicAdd(rw_count, 64);
		//	atomicAdd(nonce_hash_count + block/64, 1);
		}
		for (u32 nonce = block - 1; alive64;)
		{						// -1 compensates for 1-based ffs
			u32 ffs = __ffsll (alive64);
			nonce += ffs;
			alive64 >>= ffs;

			u64 edge = buf[nonce - block] ^ last;
			u32 u = (edge >> (uorv ? 32 : 0)) & EDGEMASK;

			if ((u & PART_MASK) == part)
			{
				nonleaf.set (u >> PART_BITS);
			//	atomicAdd(rw_count, 1);
			}

		}
	}
}

__global__ void kill_leaf_edges (cuckoo_ctx * ctx, u32 uorv, u32 part, unsigned long long int* hash_count, int* nonce_hash_count, unsigned long long int *rw_count)
{
	shrinkingset & alive = ctx->alive;
	twice_set & nonleaf = ctx->nonleaf;
	siphash_keys sip_keys = ctx->sip_keys;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	u64 buf[64];
	for (u32 block = id * 64; block < NEDGES; block += ctx->nthreads * 64)
	{
		u64 alive64 = alive.block (block);
		u64 last = 0;
		if(alive64)
		{
			last = dipblock (sip_keys, block, buf);
		//	atomicAdd(hash_count, 64);
		//	atomicAdd(rw_count, 64);
		//	atomicAdd(nonce_hash_count + block/64, 1);
		}
		for (u32 nonce = block - 1; alive64;)
		{						// -1 compensates for 1-based ffs
			u32 ffs = __ffsll (alive64);
			nonce += ffs;
			alive64 >>= ffs;

			u64 edge = buf[nonce - block] ^ last;
			u32 u = (edge >> (uorv ? 32 : 0)) & EDGEMASK;

			if ((u & PART_MASK) == part)
			{
				if (!nonleaf.test (u >> PART_BITS))
				{
					alive.reset (nonce);
				//	atomicAdd(rw_count, 1);
				}
			}
		}
	}
}

/*
__global__ void kill_leaf_edges(cuckoo_ctx *ctx, u32 uorv, u32 part) {
  shrinkingset &alive = ctx->alive;
  twice_set &nonleaf = ctx->nonleaf;
  siphash_keys sip_keys = ctx->sip_keys;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for (u32 block = id*64; block < NEDGES; block += ctx->nthreads*64) {
    u64 alive32 = alive.block(block);
    for (u32 nonce = block-1; alive32; ) { // -1 compensates for 1-based ffs
      u32 ffs = __ffsll(alive32);
      nonce += ffs; alive32 >>= ffs;
      u32 u = dipnode(sip_keys, nonce, uorv);
      if ((u & PART_MASK) == part) {
        if (!nonleaf.test(u >> PART_BITS)) {
          alive.reset(nonce);
        }
      }
    }
  }
}
*/
u32 path (cuckoo_hash & cuckoo, word_t u, word_t * us)
{
	u32 nu;
	for (nu = 0; u; u = cuckoo[u])
	{
		if (nu >= MAXPATHLEN)
		{
			while (nu-- && us[nu] != u) ;
			if (nu == ~0)
				printf ("maximum path length exceeded\n");
			else
				printf ("illegal % 4d-cycle\n", MAXPATHLEN - nu);
			exit (0);
		}
		us[nu++] = u;
	}
	return nu - 1;
}

u64 sipblock (siphash_keys & keys, const word_t edge, u64 * buf)
{
	u64 v0 = keys.k0, v1 = keys.k1, v2 = keys.k2, v3 = keys.k3;
	word_t edge0 = edge & ~EDGE_BLOCK_MASK;
	u32 i;
	for (i = 0; i < EDGE_BLOCK_MASK; i++)
	{
		word_t nonce = edge0 + i;
		v3 ^= nonce;
		SIPROUND;
		SIPROUND;
		v0 ^= nonce;
		v2 ^= 0xff;
		SIPROUND;
		SIPROUND;
		SIPROUND;
		SIPROUND;
		buf[i] = (v0 ^ v1) ^ (v2 ^ v3);
	}
	word_t nonce = edge0 + i;
	v3 ^= nonce;
	SIPROUND;
	SIPROUND;
	v0 ^= nonce;
	v2 ^= 0xff;
	SIPROUND;
	SIPROUND;
	SIPROUND;
	SIPROUND;

	buf[i] = 0;
	return (v0 ^ v1) ^ (v2 ^ v3);
}

typedef std::pair < word_t, word_t > edge;

#include <unistd.h>

int main (int argc, char **argv)
{
	int nthreads = 16384;
	int trims = 32;
	int tpb = 0;
	int nonce = 0;
	int range = 1;
	const char *header = "";
	int c;
	while ((c = getopt (argc, argv, "h:n:m:r:t:p:")) != -1)
	{
		switch (c)
		{
		case 'h':
			header = optarg;
			break;
		case 'n':
			nonce = atoi (optarg);
			break;
		case 'm':
			trims = atoi (optarg);
			break;
		case 't':
			nthreads = atoi (optarg);
			break;
		case 'p':
			tpb = atoi (optarg);
			break;
		case 'r':
			range = atoi (optarg);
			break;
		}
	}
	if (!tpb)					// if not set, then default threads per block to roughly square root of threads
		for (tpb = 1; tpb * tpb < nthreads; tpb *= 2) ;

	printf ("Looking for %d-cycle on cuckoo%d(\"%s\",%d", PROOFSIZE, NODEBITS, header, nonce);
	if (range > 1)
		printf ("-%d", nonce + range - 1);
	printf (") with 50%% edges, %d trims, %d threads %d per block\n", trims, nthreads, tpb);

	cuckoo_ctx ctx (nthreads);

	char headernonce[HEADERLEN];
	u32 hdrlen = strlen (header);
	memcpy (headernonce, header, hdrlen);
	memset (headernonce + hdrlen, 0, sizeof (headernonce) - hdrlen);

	u64 edgeBytes = NEDGES / 8, nodeBytes = TWICE_WORDS * sizeof (u32);
	checkCudaErrors (hipMalloc ((void **) &ctx.alive.bits, edgeBytes));
	checkCudaErrors (hipMalloc ((void **) &ctx.nonleaf.bits, nodeBytes));

	int edgeUnit = 0, nodeUnit = 0;
	u64 eb = edgeBytes, nb = nodeBytes;
	for (; eb >= 1024; eb >>= 10)
		edgeUnit++;
	for (; nb >= 1024; nb >>= 10)
		nodeUnit++;
	printf ("Using %d%cB edge and %d%cB node memory.\n", (int) eb, " KMGT"[edgeUnit], (int) nb, " KMGT"[nodeUnit]);

	cuckoo_ctx *device_ctx;
	checkCudaErrors (hipMalloc ((void **) &device_ctx, sizeof (cuckoo_ctx)));

	hipEvent_t start, stop;
	checkCudaErrors (hipEventCreate (&start));
	checkCudaErrors (hipEventCreate (&stop));
	unsigned long long k0 = 0xa34c6a2bdaa03a14ULL;
	unsigned long long k1 = 0xd736650ae53eee9eULL;
	unsigned long long k2 = 0x9a22f05e3bffed5eULL;
	unsigned long long k3 = 0xb8d55478fa3a606dULL;
	ctx.sip_keys.k0 = k0;
	ctx.sip_keys.k1 = k1;
	ctx.sip_keys.k2 = k2;
	ctx.sip_keys.k3 = k3;
	printf("%lu, %lu, %lu, %lu\n", ctx.sip_keys.k0, ctx.sip_keys.k1, ctx.sip_keys.k2, ctx.sip_keys.k3);

	unsigned long hash_count = 0;
	unsigned long long int*dev_hash_count;
	hipMalloc((void**)&dev_hash_count, sizeof(unsigned long long int));
	int *nonce_hash_count = (int*)malloc(sizeof(int) * NEDGES/64);
	int *dev_nonce_hash_count;
	unsigned long long int  *dev_rw_count;
	hipMalloc((void**)&dev_nonce_hash_count, sizeof(int) * NEDGES/64);
	hipMalloc((void**)&dev_rw_count, sizeof(unsigned long long int));
	hipMemcpy(dev_rw_count, &hash_count, sizeof(unsigned long long int), hipMemcpyHostToDevice);

	FILE *fcount = fopen("cuckaroo_lean.txt", "w");
	range = 1;
	for (int r = 0; r < range; r++)
	{
		clock_t cstart = clock ();
		hipEventRecord (start, NULL);
		checkCudaErrors (hipMemset (ctx.alive.bits, 0, edgeBytes));
		ctx.setheadernonce (headernonce, nonce + r);
		hipMemcpy (device_ctx, &ctx, sizeof (cuckoo_ctx), hipMemcpyHostToDevice);
	//	trims += 1;
		for (u32 round = 0; round < trims; round++)
		{
			for (u32 uorv = 0; uorv < 2; uorv++)
			{
				for (u32 part = 0; part <= PART_MASK; part++)
				{
					checkCudaErrors (hipMemset (ctx.nonleaf.bits, 0, nodeBytes));
					count_node_deg <<< nthreads / tpb, tpb >>> (device_ctx, uorv, part, dev_hash_count, dev_nonce_hash_count, dev_rw_count);
					kill_leaf_edges <<< nthreads / tpb, tpb >>> (device_ctx, uorv, part, dev_hash_count, dev_nonce_hash_count, dev_rw_count);
				}
			}
		}

		u64 *bits;
		bits = (u64 *) calloc (NEDGES / 64, sizeof (u64));
		assert (bits != 0);
		hipMemcpy (bits, ctx.alive.bits, (NEDGES / 64) * sizeof (u64), hipMemcpyDeviceToHost);

		hipEventRecord (stop, NULL);
		hipEventSynchronize (stop);
		float duration;
		hipEventElapsedTime (&duration, start, stop);
		u32 cnt = 0;
		for (int i = 0; i < NEDGES / 64; i++)
			cnt += __builtin_popcountll (~bits[i]);
		printf ("trim result : %u\n", cnt);
		u32 load = (u32) (100L * cnt / CUCKOO_SIZE);
		hipMemcpy(&hash_count, dev_hash_count, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
//		hipMemcpy(nonce_hash_count, dev_nonce_hash_count, sizeof(int) * NEDGES/64, hipMemcpyDeviceToHost);
		unsigned long rw_count = 0;
		hipMemcpy(&rw_count, dev_rw_count, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
		printf ("nonce %d: %d trims completed in %.3f seconds final load %d%%, hash count = %lu, rw_count = %lu\n", nonce + r, trims, duration / 1000.0f, load, hash_count, rw_count);

		/*FILE*fp = fopen("nonce_hash_count2.txt", "w+");
		if(fp == NULL) return;
		for(int i = 0; i < NEDGES/64; i++){
			//for(int j = 0; j < 64; j++)
			{
			//	fprintf(fp, "%d %d\n", i*64, nonce_hash_count[i]);		
			}
		}
		fclose(fp);
		hipFree(dev_nonce_hash_count);
		hipFree(dev_hash_count);
*/
		if (load >= 90)
		{
			printf ("overloaded! exiting...");
			exit (0);
		}

		cuckoo_hash & cuckoo = *(new cuckoo_hash ());
		word_t us[MAXPATHLEN], vs[MAXPATHLEN];

		edge *trimeedges = (edge *) malloc (sizeof (edge) * cnt);
		u64 buf[64];
		for (word_t block = 0, i = 0; block < NEDGES; block += 64)
		{
			u64 alive64 = ~bits[block / 64];
			if (!alive64)
				continue;
			const u64 last = sipblock (ctx.sip_keys, block, buf);
			for (word_t nonce = block - 1; alive64;)
			{
				u32 ffs = __builtin_ffsll (alive64);
				nonce += ffs;
				if (ffs == 64) alive64 = 0;
				else
				alive64 >>= ffs;
				if (nonce - block < 64)
				{
					u64 one_edge = buf[nonce - block] ^ last;
					word_t u0 = one_edge & EDGEMASK;
					word_t v0 = (one_edge >> 32) & EDGEMASK;
					edge newedge (u0, v0);
					trimeedges[i++] = newedge;
				}
			}
		}
		printf("find cycle :\n");
		for (int i = 0; i < cnt; i++)
		{
			word_t u0 = trimeedges[i].first;	//one_edge & EDGEMASK;
			word_t v0 = trimeedges[i].second;	//(one_edge >> 32) & EDGEMASK;
			if (u0)
			{
				u32 nu = path (cuckoo, u0, us), nv = path (cuckoo, v0, vs);
				if (us[nu] == vs[nv])
				{
					u32 min = nu < nv ? nu : nv;
					for (nu -= min, nv -= min; us[nu] != vs[nv]; nu++, nv++) ;
					u32 len = nu + nv + 1;
					printf ("%4d-cycle found at %d\n", len, 0);
					if (len == PROOFSIZE)
					{
						printf ("Solution");
						std::set < edge > cycle;
						u32 n = 0;
						cycle.insert (edge (*us, *vs));
						while (nu--)
							cycle.insert (edge (us[(nu + 1) & ~1], us[nu | 1]));	// u's in even position; v's in odd
						while (nv--)
							cycle.insert (edge (vs[nv | 1], vs[(nv + 1) & ~1]));	// u's in odd position; v's in even
						u64 tmpbuf[64];
						for (int j = 0; j < cnt; j++)
						{
							edge e = trimeedges[j];
							if (cycle.find (e) != cycle.end ())
							{
								//printf(" %jx", (uintmax_t)nce);
								if (PROOFSIZE > 2)
									cycle.erase (e);
								n++;
							}
						}
						assert (n == PROOFSIZE);
						printf ("\n");
					}
				}
				else if (nu < nv)
				{
					while (nu--)
						cuckoo.set (us[nu + 1], us[nu]);
					cuckoo.set (u0, v0);
				}
				else
				{
					while (nv--)
						cuckoo.set (vs[nv + 1], vs[nv]);
					cuckoo.set (v0, u0);
				}
			}
		}
		
		clock_t cend = clock ();
		printf ("all time : %.4f\n", (double) (cend - cstart) / CLOCKS_PER_SEC);
		fprintf(fcount, "%d %d %.3f %.3f\n", trims, cnt, duration/1000.0f, (double)(cend-cstart)/CLOCKS_PER_SEC - duration/1000.0f);
		free(trimeedges);
		free(bits);
	}
	fclose(fcount);
	checkCudaErrors (hipFree (ctx.alive.bits));
	checkCudaErrors (hipFree (ctx.nonleaf.bits));
	return 0;
}
